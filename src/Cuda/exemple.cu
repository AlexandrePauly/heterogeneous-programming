#include "hip/hip_runtime.h"
// Bibliothèques importées
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "lodepng.h"  // Bibliothèque pour le chargement et la sauvegarde des images PNG
 
#define SIGMA_S 2.0   // Paramètre de lissage spatial (contrôle la diffusion dans l'espace)
#define SIGMA_R 50.0  // Paramètre de préservation des contours (contrôle la diffusion en intensité)
#define KERNEL_SIZE 5 // Taille du noyau du filtre bilatéral
 
// Structure pour stocker une image
typedef struct {
    unsigned char *image;   // Pointeur vers les pixels de l'image
    unsigned width, height; // Dimensions de l'image
} Image;
 
// Fonction pour charger une image PNG en mémoire
void load_image(const char *filename, Image *img) {
    unsigned error = lodepng_decode32_file(&img->image, &img->width, &img->height, filename);
    if (error) {
        printf("Erreur lors du chargement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}
 
// Fonction pour sauvegarder une image PNG
void save_image(const char *filename, Image *img) {
    unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);
    if (error) {
        printf("Erreur lors de l'enregistrement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}
 
// Filtre bilatéral implémenté sur GPU
__global__ void bilateral_filter_cuda(unsigned char *d_input, unsigned char *d_output, int width, int height) {
    // Calcul de la position du thread dans l'image -> un thread est assigné à chaque pixel unique de l’image.
    // Thread permet : une exécution parallèle, l'optimisation des performances et la réactivité
    // blockIdx.x et blockIdx.y : Indice du bloc actuel
    // blockDim.x et blockDim.y : Nombre de threads par bloc
    // threadIdx.x et threadIdx.y : Indice du thread dans son bloc
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
 
    // Vérifier si le thread est en dehors de l'image
    if (x >= width || y >= height) return;
 
    // Calcul de l'index du pixel actuel dans l'image (RGBA, donc 4 canaux)
    int idx = 4 * (y * width + x);
    float sum_r = 0, sum_g = 0, sum_b = 0, norm_factor = 0;
    float sigma_s2 = 2.0 * SIGMA_S * SIGMA_S; // Variance du filtre spatial
    float sigma_r2 = 2.0 * SIGMA_R * SIGMA_R; // Variance du filtre en intensité
    int half_size = KERNEL_SIZE / 2;          // Moitié de la taille du noyau
 
    // Récupération des valeurs des canaux du pixel central
    unsigned char r = d_input[idx], g = d_input[idx+1], b = d_input[idx+2];
 
    // Parcours de la fenêtre de voisinage autour du pixel actuel
    for (int i = -half_size; i <= half_size; i++) {
        for (int j = -half_size; j <= half_size; j++) {
            int yy = y + i;
            int xx = x + j;

            // Si le voisin est dans l'image
            // C'est pour cette raison, que le lissage est meilleur au centre de l'image que sur les bords.
            if (xx >= 0 && xx < width && yy >= 0 && yy < height) {
                int neighbor_idx = 4 * (yy * width + xx);
                unsigned char nr = d_input[neighbor_idx], ng = d_input[neighbor_idx+1], nb = d_input[neighbor_idx+2];
                
                // Calcul des poids spatial et d'intensité
                float intensity_diff = (r - nr) * (r - nr) + (g - ng) * (g - ng) + (b - nb) * (b - nb);
                float range_weight = expf(-intensity_diff / sigma_r2);
                float spatial_weight = expf(-(i * i + j * j) / sigma_s2);
                float weight = spatial_weight * range_weight;
                
                // Accumulation des valeurs pondérées
                sum_r += weight * nr;
                sum_g += weight * ng;
                sum_b += weight * nb;
                norm_factor += weight;
            }
        }
    }
 
    // Mise à jour des valeurs filtrées
    d_output[idx] = (unsigned char)(sum_r / norm_factor);
    d_output[idx+1] = (unsigned char)(sum_g / norm_factor);
    d_output[idx+2] = (unsigned char)(sum_b / norm_factor);
    d_output[idx+3] = d_input[idx+3]; // Conserver l'alpha
}
 
// Fonction de gestion du filtre bilatéral en CUDA
void bilateral_filter(Image *img) {
    // Initialisation des variables
    int width = img->width;   // Width de l'image
    int height = img->height; // Height de l'image
    size_t img_size = width * height * 4 * sizeof(unsigned char); // Dimensions de l'image
    unsigned char *d_input, *d_output;
    
    // Allocation mémoire sur GPU
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMemcpy(d_input, img->image, img_size, hipMemcpyHostToDevice);
 
    // Définition des tailles de bloc et de grille
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    
    // Lancement du noyau CUDA
    bilateral_filter_cuda<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize(); // Attente de la fin de l'exécution GPU
    
    // Récupération des résultats
    hipMemcpy(img->image, d_output, img_size, hipMemcpyDeviceToHost);
    
    // Libération de la mémoire GPU
    hipFree(d_input);
    hipFree(d_output);
}
 
// Programme principal
int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s input.png output.png\n", argv[0]);
        return 1;
    }
 
    Image img;
    
    // Chargement de l'image
    load_image(argv[1], &img);
    
    // Mesure du temps d'exécution sur GPU
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);  // Début du chronomètre
    
    // Application du filtre bilatéral
    bilateral_filter(&img);
    
    hipEventRecord(stop, 0);   // Fin du chronomètre
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("Temps total d'exécution du filtre bilatéral : %f ms\n", elapsedTime);
    
    // Sauvegarde de l'image traitée
    save_image(argv[2], &img);
    
    // Libération de la mémoire
    free(img.image);
    
    return 0;
}